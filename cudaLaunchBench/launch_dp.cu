#include "hip/hip_runtime.h"
/**
 * @file launch_dp.cu
 * @brief Dynamic parallelism sequence for kernels A -> B -> C -> D.
 *
 * Strategy 2: a small "parent" kernel launches the four child kernels
 * from device side, queued on the tail of the same stream so they run
 * strictly in order without device-side blocking.
 *
 * Data flow:
 *   A(out=buf0)
 *   B(in=buf0, out=buf1)
 *   C(in=buf1, out=buf0)
 *   D(in=buf0, out=buf1)  // final result in buf1
 *
 * Build requirements:
 * - Compute capability sm_35+ (dynamic parallelism).
 * - Separable compilation / device linking enabled (-rdc=true).
 *
 * company - Studio Nyx
 * Copyright (c) Studio Nyx. All rights reserved.
 */
#include "kernels.cuh"

#include <hip/hip_runtime.h>
#include <stdexcept>

namespace bench
{

/**
 * @brief Parent kernel that launches A->B->C->D from device side.
 *
 * The launches use cudaStreamTailLaunch so they are appended to the same
 * execution stream in strict order. No device-side synchronization is
 * required; nested execution guarantees children complete before parent
 * returns.
 *
 * @param buf0     Device buffer used as output of A, input of B, output of C.
 * @param buf1     Device buffer used as output of B and final output of D.
 * @param n        Number of elements.
 * @param iters    Per-thread arithmetic work scaling (>= 1).
 * @param grid     Grid size to use for all child kernels.
 * @param block    Block size to use for all child kernels.
 */
__global__ void dp_sequence_kernel(float* buf0,
                                   float* buf1,
                                   int n,
                                   int iters,
                                   dim3 grid,
                                   dim3 block)
{
    // Use a single launcher thread to avoid duplicate child launches.
    if (blockIdx.x == 0 && threadIdx.x == 0)
    {
        // A
        kernelA<<<grid, block, 0, cudaStreamTailLaunch>>>(buf0, n, iters);

        // B
        kernelB<<<grid, block, 0, cudaStreamTailLaunch>>>(buf0, buf1, n, iters);

        // C
        kernelC<<<grid, block, 0, cudaStreamTailLaunch>>>(buf1, buf0, n, iters);

        // D
        kernelD<<<grid, block, 0, cudaStreamTailLaunch>>>(buf0, buf1, n, iters);
    }
}

/**
 * @brief Host wrapper that launches the device-side sequence kernel.
 *
 * @param buf0    Device buffer used as output of A, input of B, output of C.
 * @param buf1    Device buffer used as output of B and final output of D.
 * @param n       Number of elements.
 * @param iters   Per-thread arithmetic work scaling (>= 1).
 * @param grid    Grid size for child kernels.
 * @param block   Block size for child kernels.
 * @param stream  CUDA stream for the parent kernel launch (default 0).
 *
 * Note: Synchronize the stream or use CUDA events around this call to time
 * dynamic parallelism end-to-end. The parent will not complete until all
 * children have finished (nested execution).
 */
void launch_dp_sequence(float* buf0,
                        float* buf1,
                        int n,
                        int iters,
                        dim3 grid,
                        dim3 block,
                        hipStream_t stream /*=0*/)
{
    // Launch a single-thread parent that enqueues child kernels.
    dp_sequence_kernel<<<1, 1, 0, stream>>>(buf0, buf1, n, iters, grid, block);
    (void)hipPeekAtLastError();
}

/**
 * @brief Convenience wrapper that auto-computes grid size from n and block size.
 *
 * @param buf0       Device buffer used as output of A, input of B, output of C.
 * @param buf1       Device buffer used as output of B and final output of D.
 * @param n          Number of elements.
 * @param iters      Per-thread arithmetic work scaling (>= 1).
 * @param blockSize  Threads per block for child kernels (default 256).
 * @param stream     CUDA stream for the parent kernel launch (default 0).
 */
void launch_dp_sequence_auto(float* buf0,
                             float* buf1,
                             int n,
                             int iters,
                             int blockSize /*=256*/,
                             hipStream_t stream /*=0*/)
{
    if (blockSize <= 0)
    {
        blockSize = 256;
    }
    dim3 block(static_cast<unsigned>(blockSize), 1, 1);
    dim3 grid(static_cast<unsigned>((n + blockSize - 1) / blockSize), 1, 1);
    launch_dp_sequence(buf0, buf1, n, iters, grid, block, stream);
}

} // namespace bench
