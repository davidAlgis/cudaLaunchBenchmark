#include "hip/hip_runtime.h"
/**
 * @file main.cu
 * @brief Entry point for CUDA launch benchmark: host-sequenced vs dynamic parallelism.
 *
 * This program allocates two device buffers and runs a simple pipeline of four
 * kernels A -> B -> C -> D in two ways:
 *   1) Host-side sequential launches.
 *   2) Device-side dynamic parallelism (child kernels launched from a parent).
 *
 * It measures elapsed time with CUDA events, over multiple iterations, and
 * prints per-run timings and averages.
 *
 * Build requirements:
 * - CMake with CUDA separable compilation enabled (-rdc=true) for DP.
 * - Compute capability sm_35+ for dynamic parallelism.
 *
 * Usage examples:
 *   ./cudaLaunchBench
 *   ./cudaLaunchBench --n 1000000 --iters 128 --runs 20 --block 256
 *
 * company - Studio Nyx
 * Copyright (c) Studio Nyx. All rights reserved.
 */

#include "kernels.cuh"  // kernel signatures

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <vector>
#include <string>
#include <iomanip>

// Prototypes from launchers (implemented in launch_host.cu and launch_dp.cu)
namespace bench
{
void launch_host_sequence_auto(float* buf0,
                               float* buf1,
                               int n,
                               int iters,
                               int blockSize = 256,
                               hipStream_t stream = 0);

void launch_dp_sequence_auto(float* buf0,
                             float* buf1,
                             int n,
                             int iters,
                             int blockSize = 256,
                             hipStream_t stream = 0);
} // namespace bench

/*--------------------------------------------------------------------*/
/*  Simple CLI parsing                                                */
/*--------------------------------------------------------------------*/
struct Args
{
    int n = 1 << 20;          // number of elements
    int iters = 64;           // arithmetic work per thread
    int runs = 10;            // measured repetitions per strategy
    int warmup = 3;           // warmup runs per strategy
    int block = 256;          // threads per block
    int device = 0;           // CUDA device id
};

static bool parse_int_flag(const char* flag, int& dst, int argc, char** argv)
{
    for (int i = 1; i < argc; ++i)
    {
        if (std::strcmp(argv[i], flag) == 0 && i + 1 < argc)
        {
            dst = std::atoi(argv[i + 1]);
            return true;
        }
    }
    return false;
}

static Args parse_args(int argc, char** argv)
{
    Args a;
    (void)parse_int_flag("--n", a.n, argc, argv);
    (void)parse_int_flag("--iters", a.iters, argc, argv);
    (void)parse_int_flag("--runs", a.runs, argc, argv);
    (void)parse_int_flag("--warmup", a.warmup, argc, argv);
    (void)parse_int_flag("--block", a.block, argc, argv);
    (void)parse_int_flag("--device", a.device, argc, argv);
    return a;
}

/*--------------------------------------------------------------------*/
/*  CUDA helpers                                                      */
/*--------------------------------------------------------------------*/
static void check_cuda(hipError_t e, const char* what)
{
    if (e != hipSuccess)
    {
        std::cerr << "CUDA error at " << what << ": "
                  << hipGetErrorString(e) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

static float time_with_events(std::function<void(hipStream_t)> work,
                              hipStream_t stream)
{
    hipEvent_t beg, end;
    check_cuda(hipEventCreate(&beg), "hipEventCreate(beg)");
    check_cuda(hipEventCreate(&end), "hipEventCreate(end)");
    check_cuda(hipEventRecord(beg, stream), "hipEventRecord(beg)");
    work(stream);
    check_cuda(hipEventRecord(end, stream), "hipEventRecord(end)");
    check_cuda(hipEventSynchronize(end), "hipEventSynchronize(end)");
    float ms = 0.0f;
    check_cuda(hipEventElapsedTime(&ms, beg, end), "hipEventElapsedTime");
    hipEventDestroy(beg);
    hipEventDestroy(end);
    return ms;
}

/*--------------------------------------------------------------------*/
/*  Main                                                              */
/*--------------------------------------------------------------------*/
int main(int argc, char** argv)
{
    Args args = parse_args(argc, argv);

    // Select device and report capabilities.
    int deviceCount = 0;
    check_cuda(hipGetDeviceCount(&deviceCount), "hipGetDeviceCount");
    if (deviceCount == 0)
    {
        std::cerr << "No CUDA devices found." << std::endl;
        return EXIT_FAILURE;
    }
    if (args.device < 0 || args.device >= deviceCount)
    {
        std::cerr << "Invalid --device " << args.device
                  << " (have " << deviceCount << " devices)" << std::endl;
        return EXIT_FAILURE;
    }
    check_cuda(hipSetDevice(args.device), "hipSetDevice");

    hipDeviceProp_t props{};
    check_cuda(hipGetDeviceProperties(&props, args.device), "hipGetDeviceProperties");
    const int sm = props.major * 10 + props.minor;
    const bool dp_supported = (props.major > 3) || (props.major == 3 && props.minor >= 5);

    std::cout << "Device: " << props.name
              << " (SM " << props.major << "." << props.minor << ", "
              << props.multiProcessorCount << " SMs)"
              << std::endl;
    std::cout << "Settings: n=" << args.n
              << " iters=" << args.iters
              << " runs=" << args.runs
              << " warmup=" << args.warmup
              << " block=" << args.block
              << " device=" << args.device
              << std::endl;

    // Allocate buffers.
    float* d_buf0 = nullptr;
    float* d_buf1 = nullptr;
    size_t bytes = static_cast<size_t>(args.n) * sizeof(float);
    check_cuda(hipMalloc(&d_buf0, bytes), "hipMalloc(buf0)");
    check_cuda(hipMalloc(&d_buf1, bytes), "hipMalloc(buf1)");

    // Create a stream for all work.
    hipStream_t stream = nullptr;
    check_cuda(hipStreamCreate(&stream), "hipStreamCreate");

    // Warmup host sequence.
    for (int i = 0; i < args.warmup; ++i)
    {
        bench::launch_host_sequence_auto(d_buf0, d_buf1, args.n, args.iters, args.block, stream);
    }
    check_cuda(hipStreamSynchronize(stream), "hipStreamSynchronize (host warmup)");

    // Measure host sequence.
    std::vector<float> host_ms;
    host_ms.reserve(args.runs);
    for (int i = 0; i < args.runs; ++i)
    {
        float ms = time_with_events(
            [&](hipStream_t s)
            {
                bench::launch_host_sequence_auto(d_buf0, d_buf1, args.n, args.iters, args.block, s);
            },
            stream);
        host_ms.push_back(ms);
    }

    // Warmup dynamic parallelism if supported.
    std::vector<float> dp_ms;
    bool dp_ran = false;
    if (dp_supported)
    {
        for (int i = 0; i < args.warmup; ++i)
        {
            bench::launch_dp_sequence_auto(d_buf0, d_buf1, args.n, args.iters, args.block, stream);
        }
        check_cuda(hipStreamSynchronize(stream), "hipStreamSynchronize (dp warmup)");

        // Measure DP.
        dp_ran = true;
        dp_ms.reserve(args.runs);
        for (int i = 0; i < args.runs; ++i)
        {
            float ms = time_with_events(
                [&](hipStream_t s)
                {
                    bench::launch_dp_sequence_auto(d_buf0, d_buf1, args.n, args.iters, args.block, s);
                },
                stream);
            dp_ms.push_back(ms);
        }
    }
    else
    {
        std::cout << "Dynamic parallelism not supported on this device (requires SM 3.5+)."
                  << std::endl;
    }

    // Reduce results.
    auto mean = [](const std::vector<float>& v) -> double
    {
        if (v.empty())
        {
            return 0.0;
        }
        double sum = 0.0;
        for (float x : v)
        {
            sum += static_cast<double>(x);
        }
        return sum / static_cast<double>(v.size());
    };

    auto minv = [](const std::vector<float>& v) -> double
    {
        if (v.empty())
        {
            return 0.0;
        }
        double m = v.front();
        for (float x : v)
        {
            if (x < m)
            {
                m = x;
            }
        }
        return m;
    };

    auto maxv = [](const std::vector<float>& v) -> double
    {
        if (v.empty())
        {
            return 0.0;
        }
        double m = v.front();
        for (float x : v)
        {
            if (x > m)
            {
                m = x;
            }
        }
        return m;
    };

    // Print per-run and summary.
    std::cout << std::fixed << std::setprecision(3);

    std::cout << "\nHost-sequenced runs (ms):";
    for (size_t i = 0; i < host_ms.size(); ++i)
    {
        std::cout << (i == 0 ? " " : ", ") << host_ms[i];
    }
    std::cout << "\nHost-sequenced avg/min/max (ms): "
              << mean(host_ms) << " / " << minv(host_ms) << " / " << maxv(host_ms)
              << std::endl;

    if (dp_ran)
    {
        std::cout << "\nDynamic-parallelism runs (ms):";
        for (size_t i = 0; i < dp_ms.size(); ++i)
        {
            std::cout << (i == 0 ? " " : ", ") << dp_ms[i];
        }
        std::cout << "\nDynamic-parallelism avg/min/max (ms): "
                  << mean(dp_ms) << " / " << minv(dp_ms) << " / " << maxv(dp_ms)
                  << std::endl;

        // Relative comparison.
        double avg_host = mean(host_ms);
        double avg_dp = mean(dp_ms);
        if (avg_dp > 0.0)
        {
            double ratio = avg_host / avg_dp;
            std::cout << "\nSpeed ratio (host / dp): " << ratio << "x" << std::endl;
        }
    }

    // Cleanup.
    hipStreamDestroy(stream);
    hipFree(d_buf0);
    hipFree(d_buf1);

    return 0;
}
