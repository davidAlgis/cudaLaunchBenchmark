#include "hip/hip_runtime.h"
/**
 * @file kernels.cu
 * @brief Definitions for toy CUDA kernels A, B, C, D used in launch benchmarks.
 *
 * The math is intentionally arbitrary but deterministic. The "iters" parameter
 * scales ALU work so you can observe scheduling/launch overhead differences
 * between host-side sequencing and dynamic parallelism.
 *
 * company - Studio Nyx
 * Copyright (c) Studio Nyx. All rights reserved.
 */
#include "kernels.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <cmath>

namespace bench
{
/**
 * @brief Small deterministic hash to vary inputs across threads.
 */
__device__ __forceinline__ float hash_u32(uint32_t x)
{
    // Thomas Wang style mix, mapped to [0,1).
    x ^= 61u;
    x ^= x >> 16;
    x *= 9u;
    x ^= x >> 4;
    x *= 0x27d4eb2du;
    x ^= x >> 15;
    const float uf = static_cast<float>(x) * (1.0f / 4294967296.0f);
    return uf;
}

/**
 * @brief A bit of ALU work to keep cores busy without memory pressure.
 */
__device__ __forceinline__ float work_unit(float v, int iters)
{
    // Clamp to at least one iteration to avoid being optimized out.
    const int kIters = iters > 0 ? iters : 1;
    float a = v;
    float b = v * 0.5f + 0.1234567f;
    for (int k = 0; k < kIters; ++k)
    {
        // Mix in some transcendental ops and fused multiply-adds.
        float s = __sinf(a);
        float c = __cosf(b);
        a = __fmaf_rn(a, 1.000173f, 0.6180339f) + s * 0.75f - c * 0.25f;
        b = __fdividef(b + 0.0001f, __fsqrt_rn(fabsf(a) + 1.0f));
        // Keep values in a reasonable range.
        a = fmodf(a, 3.1415926f);
        b = fmodf(b, 3.1415926f);
    }
    return a + 0.5f * b;
}

/*======================================================================
  Kernel A
======================================================================*/
__global__ void kernelA(float* out, int n, int iters)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
    {
        return;
    }
    const float seed = hash_u32(static_cast<uint32_t>(i)) * 6.2831853f; // 2*pi
    const float v = work_unit(seed + static_cast<float>(i) * 0.001f, iters);
    out[i] = v;
}

/*======================================================================
  Kernel B
======================================================================*/
__global__ void kernelB(const float* in, float* out, int n, int iters)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
    {
        return;
    }
    const float x = in[i] + 0.1f;
    const float v = work_unit(x, iters) + __fmaf_rn(x, x, 0.0f) * 0.01f;
    out[i] = v;
}

/*======================================================================
  Kernel C
======================================================================*/
__global__ void kernelC(const float* in, float* out, int n, int iters)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
    {
        return;
    }
    const float x = in[i];
    const float v = work_unit(x * 0.75f - 0.2f, iters);
    // Blend a bit of nonlinearity.
    out[i] = v + x * x * 0.005f - __sinf(x) * 0.25f;
}

/*======================================================================
  Kernel D
======================================================================*/
__global__ void kernelD(const float* in, float* out, int n, int iters)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
    {
        return;
    }
    const float x = in[i];
    const float v = work_unit(x + 0.3141592f, iters);
    // Final mapping to keep values well-behaved.
    out[i] = __fsqrt_rn(fabsf(v)) + 0.1f * x;
}

} // namespace bench
