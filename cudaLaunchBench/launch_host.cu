/**
 * @file launch_host.cu
 * @brief Host-side sequential launches for kernels A -> B -> C -> D.
 *
 * This file implements the "strategy 1" baseline: the host launches
 * four kernels one after another on the same CUDA stream.
 *
 * Usage pattern (example):
 *   // Device buffers (ping-pong):
 *   float* d_buf0; // size n
 *   float* d_buf1; // size n
 *   // ... allocate ...
 *   bench::launch_host_sequence_auto(d_buf0, d_buf1, n, iters);
 *   // After the call, results are in d_buf1.
 *
 * Notes:
 * - No device-wide synchronization is performed here; the caller should
 *   place CUDA events or synchronize the stream as needed for timing.
 * - Errors are checked with hipPeekAtLastError() after each launch.
 * - Grid/block can be user-provided or auto-computed.
 *
 * company - Studio Nyx
 * Copyright (c) Studio Nyx. All rights reserved.
 */
#include "kernels.cuh"

#include <hip/hip_runtime.h>
#include <stdexcept>

namespace bench {

/**
 * @brief Launch A->B->C->D sequentially on the given stream.
 *
 * @param buf0    Device buffer used as output of A, input of B, output of C.
 * @param buf1    Device buffer used as output of B and final output of D.
 * @param n       Number of elements.
 * @param iters   Per-thread arithmetic work scaling (>= 1).
 * @param grid    Grid size for all kernels.
 * @param block   Block size for all kernels.
 * @param stream  CUDA stream (default 0).
 *
 * The data flow is:
 *   A(out=buf0)
 *   B(in=buf0, out=buf1)
 *   C(in=buf1, out=buf0)
 *   D(in=buf0, out=buf1)  // final result in buf1
 */
void launch_host_sequence(float *buf0, float *buf1, int n, int iters, dim3 grid,
                          dim3 block, hipStream_t stream /*= 0*/) {
  // Kernel A
  kernelA<<<grid, block, 0, stream>>>(buf0, n, iters);
  (void)hipPeekAtLastError();

  // Kernel B
  kernelB<<<grid, block, 0, stream>>>(buf0, buf1, n, iters);
  (void)hipPeekAtLastError();

  // Kernel C
  kernelC<<<grid, block, 0, stream>>>(buf1, buf0, n, iters);
  (void)hipPeekAtLastError();

  // Kernel D
  kernelD<<<grid, block, 0, stream>>>(buf0, buf1, n, iters);
  (void)hipPeekAtLastError();
}

/**
 * @brief Convenience wrapper that auto-computes grid size from n and block
 * size.
 *
 * @param buf0      Device buffer used as output of A, input of B, output of C.
 * @param buf1      Device buffer used as output of B and final output of D.
 * @param n         Number of elements.
 * @param iters     Per-thread arithmetic work scaling (>= 1).
 * @param blockSize Threads per block (default 256).
 * @param stream    CUDA stream (default 0).
 */
void launch_host_sequence_auto(float *buf0, float *buf1, int n, int iters,
                               int blockSize /*=256*/,
                               hipStream_t stream /*=0*/) {
  if (blockSize <= 0) {
    blockSize = 256;
  }
  dim3 block(static_cast<unsigned>(blockSize), 1, 1);
  dim3 grid(static_cast<unsigned>((n + blockSize - 1) / blockSize), 1, 1);
  launch_host_sequence(buf0, buf1, n, iters, grid, block, stream);
}

} // namespace bench
